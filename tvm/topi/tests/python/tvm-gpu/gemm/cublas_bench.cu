/*
Compile: nvcc cublas_bench.cu -std=c++11 -arch=sm_61 -lcublas -o cublas_bench
Usage: ./cublas_bench N
*/

#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

using namespace std;

typedef signed char int8;
typedef int int32;

const char *cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "CUBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "CUBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "CUBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "CUBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "CUBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "CUBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "CUBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "CUBLAS_STATUS_INTERNAL_ERROR";
  }
  return "unknown error";
}

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline hipblasStatus_t checkCublas(hipblasStatus_t result) {
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
template <typename T>
void CPU_fill_rand(T *A, int nr_rows_A, int nr_cols_A) {
  int a = 1;

  for (int i = 0; i < nr_rows_A * nr_cols_A; i++) {
    A[i] = static_cast<T>(rand() / (float)(RAND_MAX / a));
  }
}

int main(int argc, char **argv) {
  if (argc < 2) {
    std::cout << argv[0] << " N" << std::endl;
    return 1;
  }

  int N = atoi(argv[1]);
  int repeats = 1000;

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  int8 *h_A = (int8 *)malloc(N * N * sizeof(int8));
  int8 *h_B = (int8 *)malloc(N * N * sizeof(int8));
  int32 *h_C = (int32 *)malloc(N * N * sizeof(int32));

  int8 *d_A, *d_B;
  int32 *d_C;

  CPU_fill_rand(h_A, N, N);
  CPU_fill_rand(h_B, N, N);
  CPU_fill_rand(h_C, N, N);

  checkCuda(hipMalloc(&d_A, N * N * sizeof(int8)));
  checkCuda(hipMalloc(&d_B, N * N * sizeof(int8)));
  checkCuda(hipMalloc(&d_C, N * N * sizeof(int32)));

  checkCuda(hipMemcpy(d_A, h_A, N * N * sizeof(int8), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_B, h_B, N * N * sizeof(int8), hipMemcpyHostToDevice));
  checkCuda(
      hipMemcpy(d_C, h_C, N * N * sizeof(int32), hipMemcpyHostToDevice));

  int lda, ldb, ldc, m, n, k;
  const int alf = 1;
  const int bet = 0;
  const auto *alpha = &alf;
  const auto *beta = &bet;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  auto sum = 0.0;
  for (int rep = 0; rep < repeats; rep++) {
    hipEventRecord(start, 0);

    m = n = k = lda = ldb = ldc = N;
    stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A,
                        HIP_R_8I, lda, d_B, HIP_R_8I, ldb, beta, d_C,
                        HIP_R_32I, ldc, HIP_R_32I, HIPBLAS_GEMM_DEFAULT);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
      cout << cublasGetErrorString(stat) << endl;
      exit(-1);
    }

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    elapsed /= 1000.0f;
    sum += elapsed;
  }
  long long num_flops = (long long)N * N * N * 2;
  auto GFLOPS = num_flops / (sum / repeats) / 1e9;

  cout << "int8: size " << N << " average: " << sum / repeats << " s " << GFLOPS
       << " GFLOPS" << endl;

  // Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);
  hipblasDestroy(handle);
  return 0;
}
